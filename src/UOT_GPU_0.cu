#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#include <sys/time.h>
#define CHECK(call){									\
	const hipError_t error=call; 							\
	if(error!=hipSuccess){								\
		printf("Error:%s.%d.\n",__FILE__,__LINE__);				\
		printf("Code:%d.Reason:%s.\n",error,hipGetErrorString(error));		\
		exit(-10*error);							\
	}										\
}
#define eps 1e-1

double cpuSecond(){
	struct timeval tp;
	gettimeofday(&tp,NULL);
	return ((double)tp.tv_sec+(double)tp.tv_usec*1e-6);
}

void printMV(float *uot,float *distri_row,float *distri_col,float *sum_row0,float *sum_col0,float *sum_col1,float error,const int m,const int n);

void UOT_typical(float *uot,float *distri_row,float *distri_col,float *sum_col0,float *sum_col1,const int m,const int n,const float fi);

__global__ void modify_row_factor(float *distri_row,float *sum_row,float fi);

__global__ void modify_col_factor(float *distri_col,float *sum_col,float fi);

__global__ void UOT_error(float *sum_col0,float *sum_col1,float *error);

__global__ void UOT_iterbase_rowsum(float *uot,float *sum_row,float *sum_col,int m,int n,int iter_y);

__global__ void UOT_iterbase_colsum(float *uot,float *sum_row,float *sum_col,int m,int n,int iter_y);

int main(int argc,char **argv){
	int m=atoi(argv[1]);
	int n=atoi(argv[2]);

	//test new
	float *uot_0=new float[m*n];
	float *distri_row_0=new float[m];
	float *distri_col_0=new float[n];
	float *sum_row0_0=new float[m];
	float *sum_col0_0=new float[n];
	float *sum_col0_1=new float[n];
	for(int i=0;i<m*n;i++){
		uot_0[i]=(float)rand()/(float)(RAND_MAX)*100;
	}
	for(int i=0;i<m;i++){
		distri_row_0[i]=(float)rand()/(float)(RAND_MAX)*100;
	}
	for(int i=0;i<n;i++){
		distri_col_0[i]=(float)rand()/(float)(RAND_MAX)*100;
	}
	int count_0=0;
	float error_0=1;
	float fi=0.5;
	double start=0,finish=0;//,start0,finish0,start1,finish1;
	for(int i=0;i<n;i++){
		for(int j=0;j<m;j++){
			sum_col0_0[i]+=uot_0[i+j*n];
		}
	}
	for(int i=0;i<m;i++){
		for(int j=0;j<n;j++){
			sum_row0_0[i]+=uot_0[i*n+j];
		}
	}
	//printMV(uot_0,distri_row_0,distri_col_0,sum_row0_0,sum_col0_0,sum_col0_1,error_0,m,n);
		
	//test typical
	float *uot_1=new float[m*n];
	float *distri_row_1=new float[m];
	float *distri_col_1=new float[n];
	float *sum_col1_0=new float[n];
	float *sum_col1_1=new float[n];
	for(int i=0;i<m*n;i++){
		uot_1[i]=uot_0[i];
	}
	for(int i=0;i<m;i++){
		distri_row_1[i]=distri_row_0[i];
	}
	for(int i=0;i<n;i++){
		distri_col_1[i]=distri_col_0[i];
	}
	UOT_typical(uot_1,distri_row_1,distri_col_1,sum_col1_0,sum_col1_1,m,n,fi);
	
	//test new
	float *d_uot_0,*d_distri_row_0,*d_distri_col_0,*d_sum_row0_0,*d_sum_col0_0,*d_sum_col0_1,*d_error_0;
	hipMalloc((float**)&d_uot_0,m*n*sizeof(float));
	hipMalloc((float**)&d_distri_row_0,m*sizeof(float));
	hipMalloc((float**)&d_distri_col_0,n*sizeof(float));
	hipMalloc((float**)&d_sum_row0_0,m*sizeof(float));
	hipMalloc((float**)&d_sum_col0_0,n*sizeof(float));
	hipMalloc((float**)&d_sum_col0_1,n*sizeof(float));
	hipMalloc((float**)&d_error_0,sizeof(float));
	hipMemcpy(d_uot_0,uot_0,m*n*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_distri_row_0,distri_row_0,m*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_distri_col_0,distri_col_0,n*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_sum_row0_0,sum_row0_0,m*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_sum_col0_0,sum_col0_0,n*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_sum_col0_1,sum_col0_1,n*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_error_0,&error_0,sizeof(float),hipMemcpyHostToDevice);
	//parameter
	const int temp=128;
	const int c_block_x=128;
	const int c_iter_y=8;
	dim3 c_block(c_block_x);
	//add '/4'
	dim3 c_grid((n+c_block_x-1)/c_block_x,(m+c_iter_y-1)/c_iter_y);
	const int r_block_x=128;
	const int r_block_y=2;
	const int r_iter_y=8;
	dim3 r_block(r_block_x,r_block_y);
	//add '/4'
	dim3 r_grid((n+r_block_x-1)/r_block_x,(m+r_block_y*r_iter_y-1)/(r_block_y*r_iter_y));
	double time=0;//,time0=0,time1=0;
	while(error_0>=eps){
		//printf("count_0=%d\n",count_0);
		count_0++;
		start=cpuSecond();
		hipMemset(d_error_0,0,sizeof(float));
		hipMemcpy(d_sum_col0_1,d_sum_col0_0,n*sizeof(float),hipMemcpyDeviceToDevice);	
		modify_col_factor<<<(n+temp-1)/temp,temp>>>(d_distri_col_0,d_sum_col0_0,fi);
		hipMemset(d_sum_row0_0,0,m*sizeof(float));
		UOT_iterbase_colsum<<<c_grid,c_block>>>(d_uot_0,d_sum_row0_0,d_sum_col0_0,m,n,c_iter_y);
		modify_row_factor<<<(n+temp-1)/temp,temp>>>(d_distri_row_0,d_sum_row0_0,fi);
		hipMemset(d_sum_col0_0,0,n*sizeof(float));
		UOT_iterbase_rowsum<<<r_grid,r_block>>>(d_uot_0,d_sum_row0_0,d_sum_col0_0,m,n,r_iter_y);
		UOT_error<<<(n+temp-1)/temp,temp>>>(d_sum_col0_0,d_sum_col0_1,d_error_0);
		hipDeviceSynchronize();
		finish=cpuSecond()-start;
		time+=finish;
		hipMemcpy(sum_row0_0,d_sum_row0_0,m*sizeof(float),hipMemcpyDeviceToHost);
		hipMemcpy(sum_col0_0,d_sum_col0_0,n*sizeof(float),hipMemcpyDeviceToHost);
		hipMemcpy(sum_col0_1,d_sum_col0_1,n*sizeof(float),hipMemcpyDeviceToHost);
		hipMemcpy(uot_0,d_uot_0,m*n*sizeof(float),hipMemcpyDeviceToHost);
		hipMemcpy(&error_0,d_error_0,sizeof(float),hipMemcpyDeviceToHost);
		//printf("error_0=%f\n",error_0);		
		//printMV(uot_0,distri_row_0,distri_col_0,sum_row0_0,sum_col0_0,sum_col0_1,error_0,m,n);
	}
	//double x=double(m)*double(n)*12/1e9;
	//printf("%f\t",1/(time/count_0)*x);
	printf("new_time=%f,",time/count_0);
	
	//compute error
	float error=0;
	for(int i=0;i<m*n;i++){
		error+=abs(uot_0[i]-uot_1[i]);
	}
	//printf("error=%f\n",error);
	
	return 0;
}

void printMV(float *uot,float *distri_row,float *distri_col,float *sum_row0,float *sum_col0,float *sum_col1,float error,const int m,const int n){
	printf("\n-----uot\n");
	for(int i=0;i<m;i++){
		for(int j=0;j<n;j++){
			printf("%f ",uot[i*n+j]);
		}
		printf("\n");
	}
	printf("\n-----distri_row\n");
	for(int i=0;i<m;i++){
		printf("%f ",distri_row[i]);
	}
	printf("\n-----distri_col\n");
	for(int i=0;i<n;i++){
		printf("%f ",distri_col[i]);
	}
	printf("\n-----sum_row0\n");
	for(int i=0;i<m;i++){
		printf("%f ",sum_row0[i]);
	}
	printf("\n-----sum_col0\n");
	for(int i=0;i<n;i++){
		printf("%f ",sum_col0[i]);
	}
	printf("\n-----sum_col1\n");
	for(int i=0;i<n;i++){
		printf("%f ",sum_col1[i]);
	}
	printf("\n-----error\n");
	printf("%f",error);
}

void UOT_typical(float *uot,float *distri_row,float *distri_col,float *sum_col0,float *sum_col1,const int m,const int n,const float fi){
	int count_1=0;
	float error_1=1;
	//pre col sum
	for(int i=0;i<n;i++){
		for(int j=0;j<m;j++){
			sum_col0[i]+=uot[i+j*n];
		}
	}
	clock_t start_1=clock();
	while(error_1>=eps){
		//printMV(uot,distri_row,distri_col,sum_col0,sum_col1,m,n);
		//printf("count_1=%d\n",count_1);
		count_1++;
		//copy
		for(int i=0;i<n;i++){
			sum_col1[i]=sum_col0[i];
		}
		//col 
		for(int i=0;i<n;i++){
			sum_col0[i]=pow(distri_col[i]/sum_col0[i],fi);
		}
		for(int i=0;i<n;i++){
			for(int j=0;j<m;j++){
				uot[i+j*n]*=sum_col0[i];
			}
		}
		//row
		for(int i=0;i<m;i++){
			float temp=0;
			for(int j=0;j<n;j++){
				temp+=uot[i*n+j];
			}
			temp=pow(distri_row[i]/temp,fi);
			for(int j=0;j<n;j++){
				uot[i*n+j]*=temp;
			}
		}
		//col sum
		memset(sum_col0,0,n*sizeof(float));
		for(int i=0;i<n;i++){
			for(int j=0;j<m;j++){
				sum_col0[i]+=uot[i+j*n];
			}
		}
		//compute error
		error_1=0;
		for(int i=0;i<n;i++){
			error_1+=fabs(sum_col0[i]-sum_col1[i]);
		}
		//printf("error_1=%f",error_1);
	}
	clock_t finish_1=clock();
	//printf("typical_time=%f\n",(float)(finish_1-start_1)/CLOCKS_PER_SEC*1000/count_1);
}

__global__ void modify_row_factor(float *distri_row,float *sum_row,float fi){
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	sum_row[id]=powf(distri_row[id]/sum_row[id],fi);
}

__global__ void modify_col_factor(float *distri_col,float *sum_col,float fi){
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	sum_col[id]=powf(distri_col[id]/sum_col[id],fi);
}

__global__ void UOT_error(float *sum_col0,float *sum_col1,float *error){
	int id=blockIdx.x*blockDim.x+threadIdx.x;
	int laneID=threadIdx.x%32;
	float t=abs(sum_col1[id]-sum_col0[id]);
	for(unsigned w=16;w>=1;w>>=1){
		t+=__shfl_down_sync(0xffffffff,t,w);
	}
	if(laneID==0) atomicAdd(error,t);
}

__global__ void UOT_iterbase_rowsum(float *uot,float *sum_row,float *sum_col,int m,int n,int iter_y){
	//int temp0=blockIdx.x+blockIdx.y+threadIdx.x+threadIdx.y;
	//if(temp0==0) printf("gridDim.x:%d,gridDim.y:%d,blockDim.x:%d,blockDim.y:%d\n",gridDim.x,gridDim.y,blockDim.x,blockDim.y);	
	//-----change along with blockDim.y-----//
	__shared__ float sm_col[2][128];
	__shared__ float sm_row[16];
	int col=blockDim.x*blockIdx.x+threadIdx.x;
	int row=iter_y*(blockDim.y*blockIdx.y+threadIdx.y);
	int idx=row*n+col;
	sm_col[threadIdx.y][threadIdx.x]=0;
	//-----change along with iter_y,blockDim.y-----//
	if(threadIdx.x<16&&threadIdx.y==0) sm_row[threadIdx.x]=sum_row[threadIdx.x+row];
	//printf("blockIdx.x:%d,blockIdx.y:%d,threadIdx.x:%d,threadIdx.y:%d,row:%d,col:%d,idx:%d,laneID:%d,offset:%d\n",blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y,row,col,idx,laneID,offset);
	__syncthreads();
	int sm_rowid=iter_y*threadIdx.y;
	#pragma unroll
	for(int i=0;i<iter_y;i++){
		float t=uot[idx];
		t*=sm_row[sm_rowid];
		sm_col[threadIdx.y][threadIdx.x]+=t;
		uot[idx]=t;
		sm_rowid++;
		idx+=n;
	}
	__syncthreads();
	//change along with blockDim.y
	if(threadIdx.y==0) atomicAdd(&sum_col[col],sm_col[0][threadIdx.x]+sm_col[1][threadIdx.x]);//+sm_col[2][laneID]+sm_col[3][laneID]);
}

__global__ void UOT_iterbase_colsum(float *uot,float *sum_row,float *sum_col,int m,int n,int iter_y){
	//int temp0=blockIdx.x+blockIdx.y+threadIdx.x+threadIdx.y;
	//if(temp0==0) printf("gridDim.x:%d,gridDim.y:%d,blockDim.x:%d,blockDim.y:%d\n",gridDim.x,gridDim.y,blockDim.x,blockDim.y);	
	int col=blockDim.x*blockIdx.x+threadIdx.x;
	int row=iter_y*blockIdx.y;
	int idx=row*n+col;
	int laneID=threadIdx.x%32;
	int offset=threadIdx.x/32;
	//-----change along with blockDim.x-----//
	__shared__ float sm_col0[128];
	__shared__ float sm_col1[4];
	//float *p_sum_col=&sum_col[col];
	sm_col0[threadIdx.x]=sum_col[col];
	//printf("blockIdx.x:%d,blockIdx.y:%d,threadIdx.x:%d,threadIdx.y:%d,row:%d,col:%d,idx:%d,laneID:%d,offset:%d\n",blockIdx.x,blockIdx.y,threadIdx.x,threadIdx.y,row,col,idx,laneID,offset);
	#pragma unroll
	for(int i=0;i<iter_y;i++){
		float t=uot[idx];
		t*=sm_col0[threadIdx.x];
		uot[idx]=t;
		for(unsigned w=16;w>=1;w>>=1){
			t+=__shfl_down_sync(0xffffffff,t,w);
		}
		if(laneID==0) sm_col1[offset]=t;
		__syncthreads();
		//-----change along with blockDim.x-----//
		if(threadIdx.x==0) atomicAdd(&sum_row[row],sm_col1[0]+sm_col1[1]+sm_col1[2]+sm_col1[3]);
		//if(threadIdx.x==1) atomicAdd(&sum_row[row],sm_col1[4]+sm_col1[5]+sm_col1[6]+sm_col1[7]);
		//if(threadIdx.x==2) atomicAdd(&sum_row[row],sm_col1[8]+sm_col1[9]+sm_col1[10]+sm_col1[11]);
		//if(threadIdx.x==3) atomicAdd(&sum_row[row],sm_col1[12]+sm_col1[13]+sm_col1[14]+sm_col1[15]);
		__syncthreads();
		row++;
		idx+=n;
	}
}
